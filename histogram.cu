#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "histogram.h"
#include "assist.h"

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     

#define TILE_SIZE 512
#define INTENSITY_RANGE 256

/* Switch of time counting */
#define CUDA_TIMING
#define CPU_SWITCH

unsigned char *input_gpu;
unsigned char *output_gpu;

/* Warm up kernel */
__global__ void kernel(unsigned char *input, 
                       unsigned char *output) {
        
  	int location = blockIdx.x * TILE_SIZE + threadIdx.x;
	
	output[location] = location % 255;
}

/* Processing GPU kernel */
__global__ void count_intensity(unsigned int *input,
								unsigned int size,
							    unsigned int *intensity_num) {
	
    unsigned int location = blockIdx.x * TILE_SIZE + threadIdx.x;

	if (location < (size >> 2)) {
		atomicAdd(&intensity_num[(unsigned char)(input[location] & 0xFF000000)], 1);
		atomicAdd(&intensity_num[(unsigned char)(input[location] & 0x00FF0000)], 1);
		atomicAdd(&intensity_num[(unsigned char)(input[location] & 0x0000FF00)], 1);
		atomicAdd(&intensity_num[(unsigned char)(input[location] & 0x000000FF)], 1);
	}
}

__global__ void prefixSum(unsigned int *intensity_num,
						  unsigned char *min_index) {
	
	for (int i = 1; i < INTENSITY_RANGE; ++i) {
		intensity_num[i] += intensity_num[i - 1];
		if (intensity_num[i] < intensity_num[i - 1]) {
			*min_index = i;
		}
	}
}

__global__ void probability(unsigned int *intensity_num,
						    double *intensity_pro,
						    unsigned int size,
							unsigned char *min_index) {
	unsigned int index = threadIdx.x;
	if (index < INTENSITY_RANGE) {
		intensity_pro[index] = ((double) (intensity_num[index] - intensity_num[*min_index])) / (size - intensity_num[*min_index]);
	}
}

__global__ void histo_equalized(unsigned char* input,
							    unsigned int size,
							    double *intensity_pro,
							    unsigned char *output) {

  	unsigned int location = blockIdx.x * TILE_SIZE + threadIdx.x;

    if (location < size) {
		output[location] = (unsigned char) ((INTENSITY_RANGE - 1) * 
                            intensity_pro[input[location]]);
	}
}

void histogram_gpu(unsigned char *data, 
                   unsigned int height, 
                   unsigned int width) {
                         
    /* Both are the same size (CPU/GPU). */
	int size = width * height;
	int gridSize = 1 + ((size - 1) / TILE_SIZE);
	
    unsigned int *intensity_num;
	double *intensity_pro;
	unsigned char *min_index;

	checkCuda(hipMalloc((void**) &input_gpu, size * sizeof(unsigned char)));
	checkCuda(hipMalloc((void**) &output_gpu, size * sizeof(unsigned char)));
	checkCuda(hipMalloc((void**) &intensity_num, INTENSITY_RANGE * sizeof(unsigned int)));
	checkCuda(hipMalloc((void**) &intensity_pro, INTENSITY_RANGE * sizeof(double)));
	checkCuda(hipMalloc((void**) &min_index, 1 * sizeof(double)));
		
     /* Copy data to GPU */
    checkCuda(hipMemcpy(input_gpu, 
			  data, 
			  size * sizeof(char), 
			  hipMemcpyHostToDevice));
	checkCuda(hipMemset(intensity_num, 0, INTENSITY_RANGE * sizeof(unsigned int)));
	checkCuda(hipMemset(min_index, 0, 1 * sizeof(unsigned int)));
	checkCuda(hipDeviceSynchronize());

        
     /* Execute algorithm */
	dim3 dimGrid(gridSize);
    dim3 dimBlock(TILE_SIZE);

     /* Kernel Call */
	#if defined(CUDA_TIMING)
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
    

	count_intensity<<<dimGrid, dimBlock>>>((unsigned int *)input_gpu,
										    size,
										    intensity_num);
	prefixSum<<<1, 1>>>(intensity_num, min_index);

	probability<<<1, INTENSITY_RANGE>>>(intensity_num, intensity_pro, size, min_index);

	histo_equalized<<<dimGrid, dimBlock>>>(input_gpu, size, intensity_pro, output_gpu);

	#if defined(CUDA_TIMING)
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif

	 /* Retrieve results from the GPU */
	checkCuda(hipMemcpy(data, 
						output_gpu, 
						size * sizeof(unsigned char), 
						hipMemcpyDeviceToHost));
                        
     /* Free resources and end the program */
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));
	checkCuda(hipFree(intensity_num));
	checkCuda(hipFree(intensity_pro));
	checkCuda(hipFree(min_index));
}

void histogram_gpu_warmup(unsigned char *data, 
					      unsigned int height, 
                          unsigned int width) {
                         
    /* Both are the same size (CPU/GPU). */
	int size = height*width;
	
	int gridSize = 1 + (( size - 1) / TILE_SIZE);
	
	 /* Allocate arrays in GPU memory */
	checkCuda(hipMalloc((void**) &input_gpu ,size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**) &output_gpu ,size*sizeof(unsigned char)));
	checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
				
	 /* Copy data to GPU */
	checkCuda(hipMemcpy(input_gpu, 
		   			 	data, 
			     	    size*sizeof(char), 
						hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());
        
	 /* Execute algorithm */
	dim3 dimGrid(gridSize);
	dim3 dimBlock(TILE_SIZE);
        
	kernel<<<dimGrid, dimBlock>>>(input_gpu, 
								  output_gpu);
                                             
	checkCuda(hipDeviceSynchronize());
        
	 /* Retrieve results from the GPU */
	checkCuda(hipMemcpy(data, 
						 output_gpu, 
						 size*sizeof(unsigned char), 
						 hipMemcpyDeviceToHost));
                        
    /* Free resources and end the program */
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));
}

